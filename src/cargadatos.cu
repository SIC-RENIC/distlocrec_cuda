#include "hip/hip_runtime.h"
/*
 * cargadatos.c
 *
 *  Created on: 13/01/2017
 *      Author: alfonso
 */

#include "distlocrec.h"

extern PLocalidad ploc;
extern PRecurso prec;
extern PDiccionario pdic;

void cargaArchivoLocs(char *);
void cargaArchivoRecs(char *);


int cantixtipo[CANTI_TIPO_REC];

int eml2conapo(int, int, int);
float deg2rad(float);
void cesfe2carte(float lat, float lng, float *res);


/**
* Función que carga los datos de localidades
*/
void cargaArchivoLocs(char * archlocs){

  printf("Carga Localidades\n");

  int est;
  int mun;
  int loc;

  float lat,lng;
  float latr,lngr;


  float *res2=(float *)malloc(3*sizeof(float));


  int pob;

  FILE *fh=fopen(archlocs,"r");

  PLocalidad p;

  int j=0;
  int i=0;
  while(fscanf(fh,"%d %d %d %f %f %d",&est,&mun,&loc,&lat,&lng,&pob)!=EOF){


    latr=deg2rad(lat);
    lngr=deg2rad(lng);
    cesfe2carte(latr,lngr,res2);

    p=(ploc+i);

    p->est=est;
    p->mun=mun;
    p->loc=loc;
    p->cconapo=eml2conapo(est,mun,loc);

    p->lat=lat;
    p->lng=lng;

    p->x=*(res2);
    p->y=*(res2+1);
    p->z=*(res2+2);

    p->pob=pob;

    for(j=0;j<CANTI_TIPO_REC;j++){
      p->dist[j]=M_PI;
    }

    p->id=i;
    i++;
  }

  printf("Localidades : %d\n",i);

  fclose(fh);

  free(res2);

}

/**
* Función que se encarga de cargar los recursos en la localidades
*/
void cargaArchivoRecs(char * archrecs){

  printf("Carga Recursos\n");

  int est;
  int mun;
  int loc;

  float lat,lng;
  float latr,lngr;

  char stipo[22];
  int tipo;
  int id;

  float *res2=(float *)malloc(3*sizeof(float));

  FILE *fh=fopen(archrecs,"r");

  PRecurso p;

  int i;

    for(i=0;i<CANTI_TIPO_REC;i++){
      cantixtipo[i]=0;
    }

    i=0;
  while(fscanf(fh,"%d %d %d %f %f %s %d %d",&est,&mun,&loc,&lat,&lng,stipo,&tipo,&id)!=EOF){


    latr=deg2rad(lat);
    lngr=deg2rad(lng);
    cesfe2carte(latr,lngr,res2);

    p=(prec+i);

    p->est=est;
    p->mun=mun;
    p->loc=loc;
    p->cconapo=eml2conapo(est,mun,loc);

    p->lat=lat;
    p->lng=lng;

    p->x=*(res2);
    p->y=*(res2+1);
    p->z=*(res2+2);

    strcpy(p->stipo,stipo);

    if((pdic+tipo)->nombre[0]=='\0'){
      strcpy((pdic+tipo)->nombre,stipo);
    }

    p->tipo=tipo;
    p->id=id;

    cantixtipo[tipo]++;

    p->uniq_id=i;
    i++;
  }

  printf("Recursos : %d\n",i);

  int total=0;
  for(i=0;i<CANTI_TIPO_REC;i++){
        printf("tipo:%d, %d\n",i,cantixtipo[i]);
        total+=cantixtipo[i];
      }
  //printf("Total: %d\n",total);

  free(res2);
}

/**
* Función que convierte las claves de Estado, Municipio y Localidad a la clave Conapo
*/
int eml2conapo(int e, int m, int l){
  return (e*1000+m)*10000+l;
}

/**
* Función que convierte los DEG a RAD
*/
float deg2rad(float x){
  return M_PI*x/180.00;
}

/**
* Función convierte coordenadas geográficas a coordenadas cartesianas espaciales
*/
void cesfe2carte(float lat, float lng, float *res){
  *(res)=sin(lng)*cos(lat);
  *(res+1)=cos(lng)*cos(lat);
  *(res+2)=sin(lat);
}



