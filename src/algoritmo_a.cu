#include "hip/hip_runtime.h"
/*
 * algoritmo_a.cu
 *
 *  Created on: 18/01/2017
 *      Author: alfonso
 */
#include "distlocrec.h"

void calculaDLRv2(float radio);
void alojaMemoria(void);
void liberaMemoria(void);
void imprimeResultado(float radio);

extern int cantiloc;
extern int cantirec;
extern PLocalidad ploc;
extern PRecurso prec;
extern PDiccionario pdic;

extern int cantixtipo[CANTI_TIPO_REC];

//coordenadas de las localidades
float *hloc_x;
float *hloc_y;
float *hloc_z;

float *dloc_x;
float *dloc_y;
float *dloc_z;

//coordenadas de los recursos

float *hrec_x;
float *hrec_y;
float *hrec_z;
int *hrec_uid;

float *drec_x;
float *drec_y;
float *drec_z;
int *drec_uid;

//valores resultantes del calculo

float *hdist_resultado;
float *ddist_resultado;

int *hidrec_resultado;
int *didrec_resultado;

/**
 *
 */
__global__ void calculadistLRv3(int nlocs, int nrecs, int nrecsR, int ntipo,
		int offset, int tambrec, float* dloc_x, float* dloc_y, float* dloc_z,
		float* drec_x, float* drec_y, float* drec_z, int* drec_uid,
		float *ddist_resultado, int *didrec_resultado) {

	extern __shared__ float s[];
	float* shfrec_x = s;
	float* shfrec_y = (s + tambrec);
	float* shfrec_z = (s + 2 * tambrec);
	int* shirec_uid = (int *) (s + 3 * tambrec);

	int id = blockIdx.x * blockDim.x + threadIdx.x;

	float dist = 4;
	int idrec = -1;
	float daux;
	int j;

	if (id < nlocs) {
		float x = *(dloc_x + id);
		float y = *(dloc_y + id);
		float z = *(dloc_z + id);

		for (j = threadIdx.x; j < tambrec; j += blockDim.x) {

			if (j < nrecsR) {
				*(shfrec_x + j) = *(drec_x + j);
				*(shfrec_y + j) = *(drec_y + j);
				*(shfrec_z + j) = *(drec_z + j);
				*(shirec_uid + j) = *(drec_uid + j);
			}

		}
		__syncthreads();

		for (int k = 0; k < tambrec && k < nrecsR; k++) {
			daux = *(shfrec_x + k) * x + *(shfrec_y + k) * y
					+ *(shfrec_z + k) * z;
			daux = acosf(daux);
			if (daux < dist) {
				dist = daux;
				idrec = *(shirec_uid + k);
			}
		}

		__syncthreads();

		*(ddist_resultado + (id * CANTI_TIPO_REC) + ntipo) = dist;
		*(didrec_resultado + (id * CANTI_TIPO_REC) + ntipo) = idrec;

	}

}

/**
 *
 */
void calculaDLRv2(float radio) {

	int tema;

	int canti_hilos = 640;
	int canti_bloques = ceil(cantiloc / canti_hilos) + 1;

	int tambrec = 3072;

	size_t tamsharedmem = sizeof(float) * 4 * tambrec;
	int cantidad_rec_alojados;

	alojaMemoria();
	hipStream_t stream[CANTI_TIPO_REC];

	int offset = 0;

	hipMemcpy(dloc_x, hloc_x, cantiloc * sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(dloc_y, hloc_y, cantiloc * sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(dloc_z, hloc_z, cantiloc * sizeof(float),
			hipMemcpyHostToDevice);

	hipMemcpy(drec_x, hrec_x, cantirec * sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(drec_y, hrec_y, cantirec * sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(drec_z, hrec_z, cantirec * sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(drec_uid, hrec_uid, cantirec * sizeof(int),
			hipMemcpyHostToDevice);

	for (tema = 0; tema < CANTI_TIPO_REC; tema++) {

		hipStreamCreate(&stream[tema]);

		if (cantixtipo[tema] > tambrec) {
			cantidad_rec_alojados = tambrec - (cantixtipo[tema] % tambrec)
					+ cantixtipo[tema];
		} else {
			cantidad_rec_alojados = cantixtipo[tema];
			tambrec = cantidad_rec_alojados;
			tamsharedmem = sizeof(float) * 4 * tambrec;
		}

		calculadistLRv3<<<canti_bloques, canti_hilos, tamsharedmem, stream[tema]>>>(
				cantiloc, cantidad_rec_alojados, cantixtipo[tema], tema, offset,
				tambrec, dloc_x, dloc_y, dloc_z, (drec_x + offset),
				(drec_y + offset), (drec_z + offset), (drec_uid + offset),
				ddist_resultado, didrec_resultado);

		offset += cantixtipo[tema];
	}

	for (tema = 0; tema < CANTI_TIPO_REC; tema++) {
		hipStreamSynchronize(stream[tema]);
	}

	hipMemcpy(hdist_resultado, ddist_resultado,
			cantiloc * CANTI_TIPO_REC * sizeof(float), hipMemcpyDeviceToHost);

	hipMemcpy(hidrec_resultado, didrec_resultado,
			cantiloc * CANTI_TIPO_REC * sizeof(int), hipMemcpyDeviceToHost);

	imprimeResultado(radio);

	liberaMemoria();
}

/**
 *
 */
void alojaMemoria(void) {
//Localidades
	hloc_x = (float*) malloc(sizeof(float) * cantiloc);
	hloc_y = (float*) malloc(sizeof(float) * cantiloc);
	hloc_z = (float*) malloc(sizeof(float) * cantiloc);

	for (int i = 0; i < cantiloc; i++) {
		Localidad p = *(ploc + i);
		*(hloc_x + i) = p.x;
		*(hloc_y + i) = p.y;
		*(hloc_z + i) = p.z;
	}

	hipMalloc((void**) &(dloc_x), cantiloc * sizeof(float));
	hipMalloc((void**) &(dloc_y), cantiloc * sizeof(float));
	hipMalloc((void**) &(dloc_z), cantiloc * sizeof(float));

//Recursos
	hrec_x = (float*) malloc(sizeof(float) * cantirec);
	hrec_y = (float*) malloc(sizeof(float) * cantirec);
	hrec_z = (float*) malloc(sizeof(float) * cantirec);
	hrec_uid = (int*) malloc(sizeof(float) * cantirec);

	for (int i = 0; i < cantirec; i++) {
		Recurso p = *(prec + i);
		*(hrec_x + i) = p.x;
		*(hrec_y + i) = p.y;
		*(hrec_z + i) = p.z;
		*(hrec_uid + i) = p.uniq_id;
	}

	hipMalloc((void**) &(drec_x), cantirec * sizeof(float));
	hipMalloc((void**) &(drec_y), cantirec * sizeof(float));
	hipMalloc((void**) &(drec_z), cantirec * sizeof(float));
	hipMalloc((void**) &(drec_uid), cantirec * sizeof(int));

//Resultados
	hipHostMalloc((void **) &(hdist_resultado),
			cantiloc * CANTI_TIPO_REC * sizeof(float));

	hipMalloc((void**) &(ddist_resultado),
			cantiloc * CANTI_TIPO_REC * sizeof(float));

	hipHostMalloc((void **) &(hidrec_resultado),
			cantiloc * CANTI_TIPO_REC * sizeof(int));

	hipMalloc((void**) &(didrec_resultado),
			cantiloc * CANTI_TIPO_REC * sizeof(int));

}

/**
 *
 */
void liberaMemoria(void) {

	hipFree(didrec_resultado);
	hipHostFree(hidrec_resultado);

	hipFree(ddist_resultado);
	hipHostFree(hdist_resultado);

	hipFree(drec_uid);
	hipFree(drec_z);
	hipFree(drec_y);
	hipFree(drec_x);

	free(hrec_uid);
	free(hrec_z);
	free(hrec_y);
	free(hrec_x);

	hipFree(dloc_z);
	hipFree(dloc_y);
	hipFree(dloc_x);

	free(hloc_z);
	free(hloc_y);
	free(hloc_x);
}

/**
 *
 */
void imprimeResultado(float radio) {
	FILE * fh;

	fh = fopen("./salidav3.txt", "w");
	for (int i = 0; i < cantiloc; i++) {
		PLocalidad pl = (ploc + i);
		for (int tema = 0; tema < CANTI_TIPO_REC; tema++) {
			float distancia = *(hdist_resultado + (i * CANTI_TIPO_REC) + tema);
			int j = *(hidrec_resultado + (i * CANTI_TIPO_REC) + tema);
			PRecurso pr = (prec + j);

			fprintf(fh, "%d,%d,%d,%s,%d,%lf,%d,%d,%d,0\n", pl->est, pl->mun,
					pl->loc, (pdic + tema)->nombre, pl->pob, radio * distancia,
					pr->est, pr->mun, pr->loc);
		}
	}

	fclose(fh);
}
