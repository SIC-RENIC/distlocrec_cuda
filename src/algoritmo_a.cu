#include "hip/hip_runtime.h"
/*
 * algoritmo_a.cu
 *
 *  Created on: 18/01/2017
 *      Author: alfonso
 */
#include "distlocrec.h"

void calculaDLRv2(float radio);
void alojaMemoria(void);
void liberaMemoria(void);
void imprimeResultado(float radio);

extern int cantiloc;
extern int cantirec;
extern PLocalidad ploc;
extern PRecurso prec;
extern PDiccionario pdic;

extern int cantixtipo[CANTI_TIPO_REC];

//coordenadas de las localidades
float *hloc_x;
float *hloc_y;
float *hloc_z;

float *dloc_x;
float *dloc_y;
float *dloc_z;

//coordenadas de los recursos

float *hrec_x;
float *hrec_y;
float *hrec_z;

float *drec_x;
float *drec_y;
float *drec_z;

//valores resultantes del calculo

float *hdist_resultado;
float *ddist_resultado;

int *hidrec_resultado;
int *didrec_resultado;

/**
 *
 */
__global__ void calculadistLRv2(int nlocs, int nrecs, int ntipo,int offset, float* dloc_x,
		float* dloc_y, float* dloc_z, float* drec_x, float* drec_y,
		float* drec_z, float *ddist_resultado, int *didrec_resultado) {

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	float dist = 4;
	int idrec = 0;
	float daux;

	if (id < nlocs) {
		float x = *(dloc_x + id);
		float y = *(dloc_y + id);
		float z = *(dloc_z + id);

		for (int j = 0; j < nrecs; j++) {
			daux = *(drec_x + j) * x + *(drec_y + j) * y + *(drec_z + j) * z;
			daux = acos(daux);
			if (daux < dist) {
				dist = daux;
				idrec = j;
			}
		}

		*(ddist_resultado + (id * CANTI_TIPO_REC) + ntipo) = dist;
		*(didrec_resultado + (id * CANTI_TIPO_REC) + ntipo) = idrec+offset;
	}

}

/**
 *
 */
void calculaDLRv2(float radio) {

	int tema;

	int canti_hilos = 640;
	int canti_bloques = ceil(cantiloc / canti_hilos) + 1;

	alojaMemoria();
	hipStream_t stream[CANTI_TIPO_REC];

	int offset = 0;

	hipMemcpy(dloc_x, hloc_x, cantiloc * sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(dloc_y, hloc_y, cantiloc * sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(dloc_z, hloc_z, cantiloc * sizeof(float),
			hipMemcpyHostToDevice);

	hipMemcpy(drec_x, hrec_x, cantirec * sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(drec_y, hrec_y, cantirec * sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(drec_z, hrec_z, cantirec * sizeof(float),
			hipMemcpyHostToDevice);

	for (tema = 0; tema < CANTI_TIPO_REC; tema++) {

		hipStreamCreate(&stream[tema]);

		calculadistLRv2<<<canti_bloques, canti_hilos, 0, stream[tema]>>>(
				cantiloc, cantixtipo[tema], tema,offset, dloc_x, dloc_y, dloc_z,
				(drec_x + offset), (drec_y + offset), (drec_z + offset),
				ddist_resultado, didrec_resultado);

		offset += cantixtipo[tema];
	}

	for (tema = 0; tema < CANTI_TIPO_REC; tema++) {
		hipStreamSynchronize(stream[tema]);
	}

	hipMemcpy(hdist_resultado, ddist_resultado,
			cantiloc * CANTI_TIPO_REC * sizeof(float), hipMemcpyDeviceToHost);

	hipMemcpy(hidrec_resultado, didrec_resultado,
			cantiloc * CANTI_TIPO_REC * sizeof(int), hipMemcpyDeviceToHost);

	imprimeResultado(radio);

	liberaMemoria();
}

/**
 *
 */
void alojaMemoria(void) {
//Localidades
	hloc_x = (float*) malloc(sizeof(float) * cantiloc);
	hloc_y = (float*) malloc(sizeof(float) * cantiloc);
	hloc_z = (float*) malloc(sizeof(float) * cantiloc);

	for (int i = 0; i < cantiloc; i++) {
		Localidad p = *(ploc + i);
		*(hloc_x + i) = p.x;
		*(hloc_y + i) = p.y;
		*(hloc_z + i) = p.z;
	}

	hipMalloc((void**) &(dloc_x), cantiloc * sizeof(float));
	hipMalloc((void**) &(dloc_y), cantiloc * sizeof(float));
	hipMalloc((void**) &(dloc_z), cantiloc * sizeof(float));

//Recursos
	hrec_x = (float*) malloc(sizeof(float) * cantirec);
	hrec_y = (float*) malloc(sizeof(float) * cantirec);
	hrec_z = (float*) malloc(sizeof(float) * cantirec);

	for (int i = 0; i < cantirec; i++) {
		Recurso p = *(prec + i);
		*(hrec_x + i) = p.x;
		*(hrec_y + i) = p.y;
		*(hrec_z + i) = p.z;

	}

	hipMalloc((void**) &(drec_x), cantirec * sizeof(float));
	hipMalloc((void**) &(drec_y), cantirec * sizeof(float));
	hipMalloc((void**) &(drec_z), cantirec * sizeof(float));

//Resultados
	hipHostMalloc((void **) &(hdist_resultado),
			cantiloc * CANTI_TIPO_REC * sizeof(float));

	hipMalloc((void**) &(ddist_resultado),
			cantiloc * CANTI_TIPO_REC * sizeof(float));

	hipHostMalloc((void **) &(hidrec_resultado),
			cantiloc * CANTI_TIPO_REC * sizeof(int));

	hipMalloc((void**) &(didrec_resultado),
			cantiloc * CANTI_TIPO_REC * sizeof(int));

}

/**
 *
 */
void liberaMemoria(void) {

	hipFree(didrec_resultado);
	hipHostFree(hidrec_resultado);

	hipFree(ddist_resultado);
	hipHostFree(hdist_resultado);

	hipFree(drec_z);
	hipFree(drec_y);
	hipFree(drec_x);

	free(hrec_z);
	free(hrec_y);
	free(hrec_x);

	hipFree(dloc_z);
	hipFree(dloc_y);
	hipFree(dloc_x);

	free(hloc_z);
	free(hloc_y);
	free(hloc_x);
}

/**
 *
 */
void imprimeResultado(float radio) {
	FILE * fh;

	fh = fopen("./salidav2.txt", "w");
	for (int i = 0; i < cantiloc; i++) {
		PLocalidad pl = (ploc + i);
		for (int tema = 0; tema < CANTI_TIPO_REC; tema++) {
			float distancia = *(hdist_resultado + (i * CANTI_TIPO_REC) + tema);
			int j = *(hidrec_resultado + (i * CANTI_TIPO_REC) + tema);
			PRecurso pr = (prec + j);

			fprintf(fh, "%d,%d,%d,%s,%d,%lf,%d,%d,%d,0\n", pl->est, pl->mun,
					pl->loc, (pdic + tema)->nombre, pl->pob, radio * distancia,
					pr->est, pr->mun, pr->loc);

		}
	}

	fclose(fh);
}
