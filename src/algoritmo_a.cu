#include "hip/hip_runtime.h"
/*
 * algoritmo_a.cu
 *
 *  Created on: 18/01/2017
 *      Author: alfonso
 */
#include "distlocrec.h"

void calculaDLRv2(float radio);
void alojaMemoria(void);
void liberaMemoria(void);
void imprimeResultado(float radio);

extern int cantiloc;
extern int cantirec;
extern PLocalidad ploc;
extern PRecurso prec;
extern PDiccionario pdic;

extern int cantixtipo[CANTI_TIPO_REC];

//coordenadas de las localidades
float *hloc_x;
float *hloc_y;
float *hloc_z;

float *dloc_x;
float *dloc_y;
float *dloc_z;

//coordenadas de los recursos

float *hrec_x;
float *hrec_y;
float *hrec_z;
int *hrec_uid;

float *drec_x;
float *drec_y;
float *drec_z;
int *drec_uid;

//valores resultantes del calculo

float *hdist_resultado[CANTI_TIPO_REC];
float *ddist_resultado[CANTI_TIPO_REC];

int *hidrec_resultado[CANTI_TIPO_REC];
int *didrec_resultado[CANTI_TIPO_REC];

/**
 *
 */
//se elimina ntipo y offset
__global__ void calculadistLRv3(int nlocs, int nrecsR, int tambrec,
		float* dloc_x, float* dloc_y, float* dloc_z, float* drec_x,
		float* drec_y, float* drec_z, int* drec_uid, float *ddist_resultado,
		int *didrec_resultado) {

	extern __shared__ float s[];
	float* shfrec_x = s;
	float* shfrec_y = (s + tambrec);
	float* shfrec_z = (s + 2 * tambrec);
	int* shirec_uid = (int *) (s + 3 * tambrec);

	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;

	float dist = 4;
	int idrec = -1;
	float daux;
	unsigned int j;

	if (id < nlocs) {
		float x = *(dloc_x + id);
		float y = *(dloc_y + id);
		float z = *(dloc_z + id);

		for (j = threadIdx.x; j < tambrec; j += blockDim.x) {

			*(shfrec_x + j) = *(drec_x + j);
			*(shfrec_y + j) = *(drec_y + j);
			*(shfrec_z + j) = *(drec_z + j);
			*(shirec_uid + j) = *(drec_uid + j);
		}

		__syncthreads();

		for (int k = 0; k < tambrec; k++) {
			daux = *(shfrec_x + k) * x + *(shfrec_y + k) * y
					+ *(shfrec_z + k) * z;
			daux = acosf(daux);
			if (daux < dist) {
				dist = daux;
				idrec = *(shirec_uid + k);
			}
		}

		__syncthreads();

		*(ddist_resultado + id) = dist;
		*(didrec_resultado + id) = idrec;

	}

}

/**
 *
 */
//se elimina ntipo y offset
__global__ void calculadistLRv3G(int nlocs, int nrecsR, int tambrec,
		float* dloc_x, float* dloc_y, float* dloc_z, float* drec_x,
		float* drec_y, float* drec_z, int* drec_uid, float *ddist_resultado,
		int *didrec_resultado) {

	extern __shared__ float s[];
	float* shfrec_x = s;
	float* shfrec_y = (s + tambrec);
	float* shfrec_z = (s + 2 * tambrec);
	int* shirec_uid = (int *) (s + 3 * tambrec);

	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;

	float dist = 4;
	int idrec = -1;
	float daux;
	unsigned int k, j, tambrec_loc = tambrec;
	int rec_faltantes = nrecsR, avance = 0;

	if (id < nlocs) {
		float x = *(dloc_x + id);
		float y = *(dloc_y + id);
		float z = *(dloc_z + id);

		do {
			j = threadIdx.x;
			while (j < tambrec_loc && (j + avance) < nrecsR) {

				*(shfrec_x + j) = *(drec_x + j + avance);
				*(shfrec_y + j) = *(drec_y + j + avance);
				*(shfrec_z + j) = *(drec_z + j + avance);
				*(shirec_uid + j) = *(drec_uid + j + avance);

				j += blockDim.x;
			}

			__syncthreads();

			for (k = 0; k < tambrec_loc; k++) {
				daux = *(shfrec_x + k) * x + *(shfrec_y + k) * y
						+ *(shfrec_z + k) * z;
				daux = acosf(daux);
				if (daux < dist) {
					dist = daux;
					idrec = *(shirec_uid + k);
				}
			}

			avance += tambrec_loc;
			rec_faltantes -= avance;

			if (rec_faltantes < tambrec_loc) {
				tambrec_loc = rec_faltantes;
			}

			__syncthreads();

		} while (rec_faltantes > 0);

		*(ddist_resultado + id) = dist;
		*(didrec_resultado + id) = idrec;

	}

}

/**
 * Kernel para pruebas en vacio no realiza ningun calculo
 */
//se elimina ntipo y offset
__global__ void calculadistLRv3G_vacio(int nlocs, int nrecsR, int tambrec,
		float* dloc_x, float* dloc_y, float* dloc_z, float* drec_x,
		float* drec_y, float* drec_z, int* drec_uid, float *ddist_resultado,
		int *didrec_resultado) {

	unsigned int id = blockIdx.x * blockDim.x + threadIdx.x;

	*(ddist_resultado + id) = 0;
	*(didrec_resultado + id) = 0;

}

/**
 *
 */
void calculaDLRv2(float radio) {

	int tema;

	int canti_hilos = 640;

	int canti_bloques = ceil(cantiloc / canti_hilos) + 1;

	//const unsigned int tambrec = 3072;
	const unsigned int tambrec = 2000;

	size_t tamsharedmem;

	alojaMemoria();
	hipStream_t stream[CANTI_TIPO_REC];

	int offset = 0;

	hipMemcpy(dloc_x, hloc_x, cantiloc * sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(dloc_y, hloc_y, cantiloc * sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(dloc_z, hloc_z, cantiloc * sizeof(float),
			hipMemcpyHostToDevice);

	hipMemcpy(drec_x, hrec_x, cantirec * sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(drec_y, hrec_y, cantirec * sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(drec_z, hrec_z, cantirec * sizeof(float),
			hipMemcpyHostToDevice);
	hipMemcpy(drec_uid, hrec_uid, cantirec * sizeof(int),
			hipMemcpyHostToDevice);

	for (tema = 0; tema < CANTI_TIPO_REC; tema++) {

		hipStreamCreate(&stream[tema]);

		if (cantixtipo[tema] < tambrec) {

			tamsharedmem = sizeof(float) * 4 * cantixtipo[tema];

			//printf("Kernel chico: %d => %d\n", tema, cantixtipo[tema]);

			calculadistLRv3<<<canti_bloques, canti_hilos, tamsharedmem,
					stream[tema]>>>(cantiloc, cantixtipo[tema],
					cantixtipo[tema], dloc_x, dloc_y, dloc_z, (drec_x + offset),
					(drec_y + offset), (drec_z + offset), (drec_uid + offset),
					ddist_resultado[tema], didrec_resultado[tema]);

		} else {
			tamsharedmem = sizeof(float) * 4 * tambrec;
			//printf("Kernel GRANDE: %d => %d\n", tema, cantixtipo[tema]);

			calculadistLRv3G<<<canti_bloques, canti_hilos, tamsharedmem,
					stream[tema]>>>(cantiloc, cantixtipo[tema], tambrec, dloc_x,
					dloc_y, dloc_z, (drec_x + offset), (drec_y + offset),
					(drec_z + offset), (drec_uid + offset),
					ddist_resultado[tema], didrec_resultado[tema]);

		}

		hipMemcpyAsync(hdist_resultado[tema], ddist_resultado[tema],
				cantiloc * sizeof(float), hipMemcpyDeviceToHost, stream[tema]);

		hipMemcpyAsync(hidrec_resultado[tema], didrec_resultado[tema],
				cantiloc * sizeof(int), hipMemcpyDeviceToHost, stream[tema]);

		offset += cantixtipo[tema];
	}

	for (tema = 0; tema < CANTI_TIPO_REC; tema++) {
		hipStreamSynchronize(stream[tema]);
	}

	imprimeResultado(radio);

	liberaMemoria();
}

/**
 *
 */
void alojaMemoria(void) {
//Localidades
	hloc_x = (float*) malloc(sizeof(float) * cantiloc);
	hloc_y = (float*) malloc(sizeof(float) * cantiloc);
	hloc_z = (float*) malloc(sizeof(float) * cantiloc);

	for (int i = 0; i < cantiloc; i++) {
		Localidad p = *(ploc + i);
		*(hloc_x + i) = p.x;
		*(hloc_y + i) = p.y;
		*(hloc_z + i) = p.z;
	}

	hipMalloc((void**) &(dloc_x), cantiloc * sizeof(float));
	hipMalloc((void**) &(dloc_y), cantiloc * sizeof(float));
	hipMalloc((void**) &(dloc_z), cantiloc * sizeof(float));

//Recursos
	hrec_x = (float*) malloc(sizeof(float) * cantirec);
	hrec_y = (float*) malloc(sizeof(float) * cantirec);
	hrec_z = (float*) malloc(sizeof(float) * cantirec);
	hrec_uid = (int*) malloc(sizeof(float) * cantirec);

	for (int i = 0; i < cantirec; i++) {
		Recurso p = *(prec + i);
		*(hrec_x + i) = p.x;
		*(hrec_y + i) = p.y;
		*(hrec_z + i) = p.z;
		*(hrec_uid + i) = p.uniq_id;
	}

	hipMalloc((void**) &(drec_x), cantirec * sizeof(float));
	hipMalloc((void**) &(drec_y), cantirec * sizeof(float));
	hipMalloc((void**) &(drec_z), cantirec * sizeof(float));
	hipMalloc((void**) &(drec_uid), cantirec * sizeof(int));

//Resultados

	for (int i = 0; i < CANTI_TIPO_REC; i++) {
		hipHostMalloc((void **) &(hdist_resultado[i]),
				cantiloc * sizeof(float));

		hipMalloc((void**) &(ddist_resultado[i]), cantiloc * sizeof(float));

		hipHostMalloc((void **) &(hidrec_resultado[i]),
				cantiloc * sizeof(int));

		hipMalloc((void**) &(didrec_resultado[i]), cantiloc * sizeof(int));
	}

}

/**
 *
 */
void liberaMemoria(void) {

	for (int i = 0; i < CANTI_TIPO_REC; i++) {
		hipFree(didrec_resultado[i]);
		hipHostFree(hidrec_resultado[i]);

	}

	hipFree(ddist_resultado);
	hipHostFree(hdist_resultado);

	hipFree(drec_uid);
	hipFree(drec_z);
	hipFree(drec_y);
	hipFree(drec_x);

	free(hrec_uid);
	free(hrec_z);
	free(hrec_y);
	free(hrec_x);

	hipFree(dloc_z);
	hipFree(dloc_y);
	hipFree(dloc_x);

	free(hloc_z);
	free(hloc_y);
	free(hloc_x);
}

/**
 *
 */
void imprimeResultado(float radio) {
	FILE * fh;

	fh = fopen("/devel/salidav4.txt", "w");
	for (int i = 0; i < cantiloc; i++) {
		PLocalidad pl = (ploc + i);
		for (int tema = 0; tema < CANTI_TIPO_REC; tema++) {
			float distancia = *(hdist_resultado[tema] + i);
			int j = *(hidrec_resultado[tema] + i);
			PRecurso pr = (prec + j);

			fprintf(fh, "%d,%d,%d,%s,%d,%lf,%d,%d,%d,0\n", pl->est, pl->mun,
					pl->loc, (pdic + tema)->nombre, pl->pob, radio * distancia,
					pr->est, pr->mun, pr->loc);
		}
	}

	fclose(fh);

}
